#include "hip/hip_runtime.h"
#include "hmm.cuh"

using namespace troy;
using namespace std;

Hmm::Hmm(CKKSEncoder* encoder, Evaluator* evaluator, RelinKeys relin_keys,
         GaloisKeys galois_keys, size_t scale)
    : encoder(encoder),
      evaluator(evaluator),
      relin_keys(relin_keys),
      galois_keys(galois_keys),
      scale(scale) {
    slot_count = encoder->slot_count();
}

void Hmm::matMult(Ciphertext a, Ciphertext b, Ciphertext& c, int m, int l,
                  int n) {
    if (l < n) {

        Ciphertext a_;
        for (size_t i = 0; i < l; i++) {
            Plaintext mask_pt;
            vector<complex<double>> mask(slot_count, 0);
            generate_lmask(i, mask, m, n);
            encoder->encode_complex64_simd(mask, a.parms_id(), scale, mask_pt);

            if (i == 0) {
                evaluator->multiply_plain(a, mask_pt, a_);
                evaluator->rescale_to_next_inplace(a_);
            } else {
                Ciphertext tmp;
                evaluator->multiply_plain(a, mask_pt, tmp);
                evaluator->rescale_to_next_inplace(tmp);
                int step = i - m * n * i;
                if (abs(step) >= slot_count / 2) {
                    step = step > 0 ? step - slot_count : slot_count + step;
                }
                if (step != 0) {
                    evaluator->rotate_vector_inplace(tmp, step, galois_keys);
                }
                evaluator->add_inplace(a_, tmp);
            }
        }

        for (size_t i = 0; i < log2(n); i++) {
            Ciphertext tmp;
            evaluator->rotate_vector(a_, -pow(2, i), galois_keys, tmp);
            evaluator->add_inplace(a_, tmp);
        }
        
        Ciphertext b_;
        for (size_t i = 0; i < l; i++) {
            Plaintext mask_pt;
            vector<complex<double>> mask(slot_count, 0);
            generate_rmask(i, mask, m, l, n);
            encoder->encode_complex64_simd(mask, b.parms_id(), scale, mask_pt);
            if (i == 0) {
                evaluator->multiply_plain(b, mask_pt, b_);
                evaluator->rescale_to_next_inplace(b_);
            } else {
                Ciphertext tmp;
                evaluator->multiply_plain(b, mask_pt, tmp);
                evaluator->rescale_to_next_inplace(tmp);
                int step = n * i - m * n * i;
                if (abs(step) > slot_count / 2) {
                    step = step > 0 ? step - slot_count : slot_count + step;
                }
                if (step != 0) {
                    evaluator->rotate_vector_inplace(tmp, step, galois_keys);
                }
                evaluator->add_inplace(b_, tmp);
            }
        }

        for (size_t i = 0; i < log2(m); i++) {
            Ciphertext tmp;
            evaluator->rotate_vector(b_, -pow(2, i) * n, galois_keys, tmp);
            evaluator->add_inplace(b_, tmp);
        }
        
        evaluator->multiply(a_, b_, c);
        evaluator->relinearize_inplace(c, relin_keys);
        evaluator->rescale_to_next_inplace(c);

        for (size_t i = 0; i < log2(l); i++) {
            Ciphertext tmp;
            int step = pow(2, i) * m * n;
            if (abs(step) >= slot_count / 2) {
                step = step > 0 ? step - slot_count : slot_count + step;
            }
            evaluator->rotate_vector(c, step, galois_keys, tmp);
            evaluator->add_inplace(c, tmp);
        }

    } else {
        
        for (size_t i = 0; i < log2(n); i++) {
            Ciphertext tmp;
            int step = -pow(2, i) * m * l;
            if (abs(step) >= slot_count / 2) {
                step = step > 0 ? step - slot_count : slot_count + step;
            }
            evaluator->rotate_vector(a, step, galois_keys, tmp);
            evaluator->add_inplace(a, tmp);
        }
        
        size_t cur_n = n;
        if (n < m) {
            while (m - cur_n >= n) {
                Ciphertext tmp;
                int step = -cur_n * l;
                if (abs(step) >= slot_count / 2) {
                    step = step > 0 ? step - slot_count : slot_count + step;
                }
                evaluator->rotate_vector(b, step, galois_keys, tmp);
                evaluator->add_inplace(b, tmp);
                cur_n *= 2;
            }
        }

        Ciphertext b_;
        for (size_t i = 0; i < n; i++) {
            vector<complex<double>> umask(slot_count, 0);
            generate_mask(umask, i, min(cur_n, i + m), l);
            Plaintext umask_pt;
            encoder->encode_complex64_simd(umask, b.parms_id(), scale,
                                           umask_pt);
            Ciphertext u_ct;
            evaluator->multiply_plain(b, umask_pt, u_ct);
            evaluator->rescale_to_next_inplace(u_ct);

            if (i == 0) {
                b_ = u_ct;
            } else {
                int step = -i * m * l + i * l;
                if (abs(step) >= slot_count / 2) {
                    step = step > 0 ? step - slot_count : slot_count + step;
                }
                if (step != 0) {
                    evaluator->rotate_vector_inplace(u_ct, step, galois_keys);
                }
                evaluator->add_inplace(b_, u_ct);
            }
            if (i + m > cur_n) {
                vector<complex<double>> vmask(slot_count, 0);
                generate_mask(vmask, 0, m + i - cur_n, l);
                Plaintext vmask_pt;
                encoder->encode_complex64_simd(vmask, std::nullopt, scale,
                                               vmask_pt);
                Ciphertext v_ct;
                evaluator->multiply_plain(b, vmask_pt, v_ct);
                evaluator->rescale_to_next_inplace(v_ct);
                int step = -i * m * l + i * l - cur_n * l;
                if (abs(step) >= slot_count / 2) {
                    step = step > 0 ? step - slot_count : slot_count + step;
                }
                if (step != 0) {
                    evaluator->rotate_vector_inplace(v_ct, step, galois_keys);
                }
                evaluator->add_inplace(b_, v_ct);
            }
        }
        
        a.scale() = scale;
        b_.scale() = scale;
        evaluator->mod_switch_to_inplace(a, b_.parms_id());
        evaluator->multiply(a, b_, c);
        evaluator->relinearize_inplace(c, relin_keys);
        evaluator->rescale_to_next_inplace(c);

        for (size_t i = 0; i < log2(l); i++) {
            Ciphertext tmp;
            evaluator->rotate_vector(c, pow(2, i), galois_keys, tmp);
            evaluator->add_inplace(c, tmp);
        }
        
        vector<complex<double>> mask(slot_count, 0);
        generate_lmask(0, mask, m * n, l);
        Plaintext lmask_pt;
        encoder->encode_complex64_simd(mask, c.parms_id(), scale, lmask_pt);
        evaluator->multiply_plain_inplace(c, lmask_pt);
        evaluator->rescale_to_next_inplace(c);

        for (size_t i = 0; i < log2(n); i++) {
            Ciphertext tmp;
            evaluator->rotate_vector(c, -pow(2, i), galois_keys, tmp);
            evaluator->add_inplace(c, tmp);
        }
        
        vector<complex<double>> filter(slot_count, 0);
        generate_filter(filter, m, n, l);
        Plaintext filter_pt;
        encoder->encode_complex64_simd(filter, c.parms_id(), scale, filter_pt);
        evaluator->multiply_plain_inplace(c, filter_pt);
        evaluator->rescale_to_next_inplace(c);
        
        for (size_t i = 0; i < log2(n); i++) {
            Ciphertext tmp;
            int step = pow(2, i) * m * l;
            if (abs(step) >= slot_count / 2) {
                step = step > 0 ? step - slot_count : slot_count + step;
            }
            evaluator->rotate_vector(c, step, galois_keys, tmp);
            evaluator->add_inplace(c, tmp);
        }
    }
}
