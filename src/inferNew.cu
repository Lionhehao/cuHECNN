#include "hip/hip_runtime.h"
#include "hmm.cuh"
#include "inferNew.cuh"
using namespace std;
using namespace troy;

void infer() {
    //////////////// Scheme Generate //////////////////
    cout << "Scheme Generate" << endl;

    EncryptionParameters parms(SchemeType::CKKS);
    size_t poly_modulus_degree = 16384;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(
        poly_modulus_degree, {45, 34, 34, 34, 34, 34, 34, 34, 34, 34, 34, 45}));
    double scale = pow(2.0, 34);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();

    if (utils::device_count() > 0) {
        context->to_device_inplace();
        encoder.to_device_inplace();
    }

    auto keyGen_start = clock();

    KeyGenerator keygen(context);
    SecretKey secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys galois_keys = keygen.create_galois_keys(false);

    auto keyGen_end = clock();
    cout << "keyGen: " << (float)(keyGen_end - keyGen_start) / CLOCKS_PER_SEC
         << "s" << endl;
    
    Encryptor encryptor(context);
    encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);

    Hmm hmm(&encoder, &evaluator, relin_keys, galois_keys, scale);
    //////////////// Encrypt Model //////////////////
    cout << "Encrypt Model" << endl;
    string weightfile = "../data/model_weights.csv";
    auto weights = readCsvToWeights(weightfile);

    auto modelEnc_start = clock();
    vector<Ciphertext> conv_cts;
    vector<Ciphertext> conv_b_cts;
    for (size_t i = 0; i < 4; i++) {
        for (size_t j = 0; j < 49; j++) {
            vector<complex<double>> conv(slot_count, 0);
            for (size_t k = 0; k < slot_count; k++) {
                conv[k] = weights[0][i * 49 + j];
            }
            Plaintext conv_pt;
            encoder.encode_complex64_simd(conv, std::nullopt, scale, conv_pt);
            Ciphertext conv_ct;
            encryptor.encrypt_asymmetric(conv_pt, conv_ct);
            conv_cts.push_back(conv_ct);
        }

        vector<complex<double>> conv_b(slot_count, 0);
        for (size_t j = 0; j < slot_count; j++) {
            conv_b[j] = weights[1][i];
        }
        Plaintext conv_b_pt;
        encoder.encode_complex64_simd(conv_b, std::nullopt, scale, conv_b_pt);
        Ciphertext conv_b_ct;
        encryptor.encrypt_asymmetric(conv_b_pt, conv_b_ct);
        conv_b_cts.push_back(conv_b_ct);
    }

    vector<Ciphertext> fct1_cts;
    Ciphertext fct1_b_ct;

    for (size_t i = 0; i < 256; i++) {
        vector<complex<double>> fct1(slot_count, 0);
        for (size_t j = 0; j < 64; j++) {
            fct1[j] = weights[2][i + j * 256];
        }
        Plaintext fct1_pt;
        encoder.encode_complex64_simd(fct1, std::nullopt, scale, fct1_pt);
        Ciphertext fct1_ct;
        encryptor.encrypt_asymmetric(fct1_pt, fct1_ct);
        fct1_cts.push_back(fct1_ct);
    }

    vector<complex<double>> fct1_b(slot_count, 0);
    for (size_t i = 0; i < 128; i++) {
        copy(weights[3].begin(), weights[3].end(), fct1_b.begin() + i * 64);
    }
    Plaintext fct1_b_pt;
    encoder.encode_complex64_simd(fct1_b, std::nullopt, scale, fct1_b_pt);
    encryptor.encrypt_asymmetric(fct1_b_pt, fct1_b_ct);

    vector<Ciphertext> fct2_cts;
    Ciphertext fct2_b_ct;

    for (size_t i = 0; i < 10; i++) {
        vector<complex<double>> fct2(slot_count, 0);
        for (size_t j = 0; j < 64; j++) {
            fct2[j] = weights[4][i * 64 + j];
        }
        Plaintext fct2_pt;
        encoder.encode_complex64_simd(fct2, std::nullopt, scale, fct2_pt);
        Ciphertext fct2_ct;
        encryptor.encrypt_asymmetric(fct2_pt, fct2_ct);
        fct2_cts.push_back(fct2_ct);
    }

    vector<complex<double>> fct2_b(slot_count, 0);
    for (size_t i = 0; i < 10; i++) {
        copy(weights[5].begin(), weights[5].end(), fct2_b.begin() + i * 64);
    }
    Plaintext fct2_b_pt;
    encoder.encode_complex64_simd(fct2_b, std::nullopt, scale, fct2_b_pt);
    encryptor.encrypt_asymmetric(fct2_b_pt, fct2_b_ct);

    auto modelEnc_end = clock();
    cout << "modelEnc: " << (float)(modelEnc_end - modelEnc_start) / CLOCKS_PER_SEC
         << "s" << endl;

    //////////////// Encrypt Data //////////////////
    cout << "Encrypt Data" << endl;
    string testfile = "../data/MNISTt10k(28x28).csv";
    auto test_datas = readCsvToTestDatas(testfile);

    auto dataEnc_start = clock();

    vector<Ciphertext> test_data_cts;
    for (size_t i = 0; i < 49; i++) {
        vector<complex<double>> test_data(slot_count, 0);
        size_t index = (i / 7) * 28 + (i % 7);
        for (size_t j = 0; j < 128; j++) {
            for (size_t k = 0; k < 64; k++) {
                test_data[j * 64 + k] =
                    (test_datas[j][index + (k / 8) * 3 * 28 + (k % 8) * 3 + 1] /
                         255.0 -
                     0.1307) /
                    0.3081;
            }
        }

        Plaintext test_data_pt;
        encoder.encode_complex64_simd(test_data, std::nullopt, scale,
                                      test_data_pt);
        Ciphertext test_data_ct;
        encryptor.encrypt_asymmetric(test_data_pt, test_data_ct);
        test_data_cts.push_back(test_data_ct);
    }

    auto dataEnc_end = clock();
    cout << "dataEnc: " << (float)(dataEnc_end - dataEnc_start) / CLOCKS_PER_SEC
         << "s" << endl;

    //////////////// Infer //////////////////
    cout << "Infer" << endl;

    auto infer_start = clock();

    vector<Ciphertext> conv_reses;
    for (size_t i = 0; i < 4; i++) {
        Ciphertext conv_res;
        for (size_t j = 0; j < 49; j++) {
            Ciphertext tmp;
            evaluator.multiply(test_data_cts[j], conv_cts[i * 49 + j], tmp);
            evaluator.relinearize_inplace(tmp, relin_keys);
            evaluator.rescale_to_next_inplace(tmp);
            if (j == 0) {
                conv_res = tmp;
            } else {
                evaluator.add_inplace(conv_res, tmp);
            }
        }
        conv_b_cts[i].scale() = scale;
        conv_res.scale() = scale;
        evaluator.mod_switch_to_inplace(conv_b_cts[i], conv_res.parms_id());
        evaluator.add_inplace(conv_res, conv_b_cts[i]);
        conv_reses.push_back(conv_res);
    }

    auto act1_start = clock();
    cout << "conv: " << (float)(act1_start - infer_start) / CLOCKS_PER_SEC
         << "s" << endl;

    for (size_t i = 0; i < 4; i++) {
        evaluator.square_inplace(conv_reses[i]);
        evaluator.relinearize_inplace(conv_reses[i], relin_keys);
        evaluator.rescale_to_next_inplace(conv_reses[i]);
    }

    auto fct1_start = clock();
    cout << "act1: " << (float)(fct1_start - act1_start) / CLOCKS_PER_SEC << "s"
         << endl;

    Ciphertext fct1_res;
    for (size_t i = 0; i < 256; i++) {
        vector<complex<double>> a_mask(slot_count, 0);
        for (size_t j = i % 64; j < 128 * 64; j += 64) {
            a_mask[j] = 1;
        }
        Plaintext a_mask_pt;
        encoder.encode_complex64_simd(a_mask, conv_reses[i / 64].parms_id(),
                                      scale, a_mask_pt);

        Ciphertext a;
        evaluator.multiply_plain(conv_reses[i / 64], a_mask_pt, a);
        evaluator.rescale_to_next_inplace(a);

        int step = i % 64;
        if (step != 0) {
            evaluator.rotate_vector_inplace(a, step, galois_keys);
        }

        evaluator.mod_switch_to_inplace(fct1_cts[i], a.parms_id());
        Ciphertext tmp;

        hmm.matMult(a, fct1_cts[i], tmp, 128, 1, 64);

        if (i == 0) {
            fct1_res = tmp;
        } else {
            evaluator.add_inplace(fct1_res, tmp);
        }
    }

    fct1_b_ct.scale() = fct1_res.scale();
    evaluator.mod_switch_to_inplace(fct1_b_ct, fct1_res.parms_id());
    evaluator.add_inplace(fct1_res, fct1_b_ct);

    auto act2_start = clock();
    cout << "fct1: " << (float)(act2_start - fct1_start) / CLOCKS_PER_SEC << "s"
         << endl;

    evaluator.square_inplace(fct1_res);
    evaluator.relinearize_inplace(fct1_res, relin_keys);
    evaluator.rescale_to_next_inplace(fct1_res);

    auto fct2_start = clock();
    cout << "act2: " << (float)(fct2_start - act2_start) / CLOCKS_PER_SEC << "s"
         << endl;

    Ciphertext fct2_res;
    for (size_t i = 0; i < 10; i++) {
        Ciphertext tmp;
        fct2_cts[i].scale() = fct1_res.scale();
        evaluator.mod_switch_to_inplace(fct2_cts[i], fct1_res.parms_id());
        hmm.matMult(fct1_res, fct2_cts[i], tmp, 128, 64, 1);

        if (i == 0) {
            fct2_res = tmp;
        } else {
            evaluator.rotate_vector_inplace(tmp, -i, galois_keys);
            evaluator.add_inplace(fct2_res, tmp);
        }
    }

    fct2_b_ct.scale() = fct2_res.scale();
    evaluator.mod_switch_to_inplace(fct2_b_ct, fct2_res.parms_id());
    evaluator.add_inplace(fct2_res, fct2_b_ct);

    auto infer_stop = clock();
    cout << "fct2: " << (float)(infer_stop - fct2_start) / CLOCKS_PER_SEC << "s"
         << endl;
    auto esp_time = (float)(infer_stop - infer_start) / CLOCKS_PER_SEC;
    printf("The time by matrixMul:\t%fs\n", esp_time);

    Plaintext res_pt;
    decryptor.decrypt(fct2_res, res_pt);
    vector<complex<double>> res;
    encoder.decode_complex64_simd(res_pt, res);
    cout << "batch result:";
    for (size_t i = 0; i < 128; i++) {
        double ans = res[i * 64].real();
        int ans_index = 0;
        for (size_t j = 1; j < 10; j++) {
            if (res[i * 64 + j].real() > ans) {
                ans = res[i * 64 + j].real();
                ans_index = j;
            }
        }
        if (i % 32 == 0) {
            cout << endl;
        }
        cout << " " << ans_index;
    }
    cout << endl;
}
