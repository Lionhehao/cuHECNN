#include "hip/hip_runtime.h"
#include "helper.cuh"
#include "hmm.cuh"
#include "inferNew.cuh"

using namespace troy;
using namespace std;

void testMatMult(int m, int l, int n) {
    EncryptionParameters parms(SchemeType::CKKS);
    size_t poly_modulus_degree = 16384;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(
        poly_modulus_degree, {60, 30, 30, 30, 30, 30, 30, 30, 60}));
    double scale = pow(2.0, 30);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();

    if (utils::device_count() > 0) {
        context->to_device_inplace();
        encoder.to_device_inplace();
    }

    KeyGenerator keygen(context);
    SecretKey secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys galois_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context);
    encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);

    vector<vector<double>> a = generate_rand_matrix<double>(m, l);
    vector<vector<double>> b = generate_rand_matrix<double>(l, n);
    vector<vector<double>> c = multiply_matrices(a, b);

    cout << "a: ";
    print_matrix(a);
    cout << "b: ";
    print_matrix(b);

    cout << "c: ";
    print_matrix(c);

    Ciphertext a_ct;
    Ciphertext b_ct;
    Ciphertext c_ct;

    if (l < n) {
        encode_matrix_l(a, n);
        vector<complex<double>> a_vec(slot_count, 0);
        pack_matrix(a, a_vec);
        Plaintext a_pt;
        encoder.encode_complex64_simd(a_vec, std::nullopt, scale, a_pt);

        encryptor.encrypt_asymmetric(a_pt, a_ct);

        vector<complex<double>> b_vec(slot_count, 0);
        pack_matrix(b, b_vec);
        Plaintext b_pt;
        encoder.encode_complex64_simd(b_vec, std::nullopt, scale, b_pt);
        encryptor.encrypt_asymmetric(b_pt, b_ct);

    } else {
        vector<complex<double>> a_vec(slot_count, 0);
        pack_matrix(a, a_vec);
        Plaintext a_pt;
        encoder.encode_complex64_simd(a_vec, std::nullopt, scale, a_pt);
        encryptor.encrypt_asymmetric(a_pt, a_ct);

        encode_matrix_r(b);
        vector<complex<double>> b_vec(slot_count, 0);
        pack_matrix(b, b_vec);
        Plaintext b_pt;
        encoder.encode_complex64_simd(b_vec, std::nullopt, scale, b_pt);
        encryptor.encrypt_asymmetric(b_pt, b_ct);
    }

    Hmm hmm(&encoder, &evaluator, relin_keys, galois_keys, scale);

    auto start = clock();
    hmm.matMult(a_ct, b_ct, c_ct, m, l, n);
    auto stop = clock();
    auto esp_time = (float)(stop - start) / CLOCKS_PER_SEC;

    Plaintext c_pt;
    decryptor.decrypt(c_ct, c_pt);
    vector<complex<double>> res;
    encoder.decode_complex64_simd(c_pt, res);
    auto matrix = convertTo2D(res, m, max(l, n));
    print_matrix(matrix);
    printf("The time by matrixMul:\t%fs\n", esp_time);
}

int main() {
    // int m, l, n;
    // while (cin >> m >> l >> n) {
    //     testMatMult(m, l, n);
    // }
    infer();
}
